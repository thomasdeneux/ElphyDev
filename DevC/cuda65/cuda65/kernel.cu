#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "CudaUtil.h"

#define PI 3.1415926536f


extern int MaxThreadsPerBlock;
extern int MaxThreadsX;
extern int MaxThreadsY;

 
__global__ void Kernel_FillGrating(unsigned char *surface, int width, int height, size_t pitch )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    char *pixel;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (x >= width || y >= height) return;

    // get a pointer to the pixel at (x,y)
    pixel = (char *)(surface + y*pitch) + 4*x;

    // populate it
	
	pixel[0] = 0;                          // red
    pixel[1] = 128 + 127*cos(2*PI/width*y);// green 
    pixel[2] = 0;                          // blue
    pixel[3] = 0;                          // alpha

}

extern "C"
void FillGrating1(void *surface, int width, int height, size_t pitch)
{
    hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);                   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_FillGrating<<<Dg,Db>>>((unsigned char *)surface, width, height, pitch );

    error = hipGetLastError();

   
}

__global__ void interpol(int z1, int z2, int z3, int z4,float dx, float dy,float* zr)
{
   float zp = z1+ dy*(z2-z1);
   float zq = z4+ dy*(z3-z4);
   *zr = zp+ dx*(zq-zp);
}

__global__ void Kernel_CartToPol1(unsigned char *surface1, unsigned char *surface2, int width, int height, size_t pitch, int BK )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (x >= width || y >= height) return;

    float ZR;
    pixel1 = (unsigned char *)(surface1 + y*pitch) + 4*x;
    if (pixel1[3] != 0) 
    {
    
    float R;
    float theta;
 

    //R = 2* sqrtf( powf(x-width/2,2) + powf(y-height/2,2) );
    //theta = (atan2f( y-height/2 ,  x-width/2) +PI)*height/(2*PI);
  
    R = 2* sqrtf( powf(x-width/2,2) + powf(y-height/2,2) );
    theta = (atan2f( y-height/2 ,  x-width/2) +PI)*height/(2*PI);
    
    if (R==0) {R=1;}
    float R2= logf(R);
    float R2max = logf(sqrtf(width*width+height*height));
    R = R2/R2max*width;
    
    int x1 = ((int) R) % width ;
    int y1 = ((int) theta) % height;

    int xp1 = (x1+1) % width;
    int yp1 = (y1+1) % height;
    
    float z1 =   *((unsigned char *)(surface1 + y1*pitch  + 4*x1+1));
    float z2 =   *((unsigned char *)(surface1 + yp1*pitch + 4*x1+1));
    float z3 =   *((unsigned char *)(surface1 + yp1*pitch + 4*xp1+1));
    float z4 =   *((unsigned char *)(surface1 + y1*pitch  + 4*xp1+1));

    float dx =  theta-floorf(theta);
    float dy =  R-floorf(R); 

    float zp = 1.0*z1+ dy*(1.0*z2-z1);
    float zq = 1.0*z4+ dy*(1.0*z3-z4);
    ZR = zp+ dx*(zq-zp);

    //if (z1<1){ z1 = 1;}
    //if (z1>=253){ z1 = 253;}
    }
    else
    { ZR = BK;} 

    pixel2 = (unsigned char *)(surface2 + y*pitch) + 4*x;
    pixel2[1] = ZR;  
   
}

void CartToPolK1(void *surface1,void *surface2, int width, int height, size_t pitch, int BK)
{
    hipError_t error = hipSuccess;

    dim3 Db = dim3(32, 32);                  
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_CartToPol1<<<Dg,Db>>>((unsigned char *)surface1,(unsigned char *)surface2, width, height, pitch, BK );

    error = hipGetLastError();

   
}

__global__ void Kernel_WaveTransformK1(unsigned char *surface1, unsigned char *surface2, int width, int height, size_t pitch,
                                       float Amp, float a, float b, float Rt, int x0, int y0, int yref, int Mask )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (x >= width || y >= height) return;

    pixel1 = (unsigned char *)(surface1 + y*pitch) + 4*x;
    pixel2 = (unsigned char *)(surface2 + y*pitch) + 4*x;
    
    float R =  sqrtf( powf(x-x0,2) + powf(y-y0,2) ) ;
    float ZR  = Amp*sin(a*R+b);
    if (Rt>0) ZR = ZR*expf(-R/Rt);
       

    for (int i=0;i<3;i++)
    {
      int w;
      if (yref>=0) w = yref + ZR;
      else
      w = pixel1[i] +ZR;

      if (w<0) w=0;
      else
      if (w>253) w=253;

      if (Mask & (1<<i))  pixel2[i] = w;  
    }
    if (yref<0)  pixel2[3] = pixel1[3]; // on copie alpha de la source
}

void WaveTransformK1( void *surface1,void *surface2, int width, int height, size_t pitch, 
                      float Amp, float a, float b, float Rt, int x0, int y0, int yref, int RgbMask )
{
    dim3 Db = dim3(32, 32);                  
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_WaveTransformK1<<<Dg,Db>>>((unsigned char *)surface1,(unsigned char *)surface2, width, height, pitch, 
                                       Amp,a,b,Rt,x0,y0,yref, RgbMask );

}


__global__ void Kernel_WaveTransformK2(unsigned char *surface1, unsigned char *surface2, int width, int height, size_t pitch,
                                       float Amp, float a, float b, int x0, int y0, int yref, int Mask )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (x >= width || y >= height) return;

    pixel1 = (unsigned char *)(surface1 + y*pitch) + 4*x;
    pixel2 = (unsigned char *)(surface2 + y*pitch) + 4*x;
    
    
    float theta = atan2f( y-y0 ,  x-x0);
    float ZR  = Amp*sin(a*theta+b);
           

    for (int i=0;i<3;i++)
    {
      int w;
      if (yref>=0) w = yref + ZR;
      else
      w = pixel1[i] +ZR;

      if (w<0) w=0;
      else
      if (w>253) w=253;

      if (Mask & (1<<i))  pixel2[i] = w;  
    }
    if (yref<0)  pixel2[3] = pixel1[3]; // on copie alpha de la source
}

void WaveTransformK2( void *surface1,void *surface2, int width, int height, size_t pitch, 
                      float Amp, float a, float b,  int x0, int y0, int yref, int RgbMask )
{
    dim3 Db = dim3(32, 32);                  
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_WaveTransformK2<<<Dg,Db>>>((unsigned char *)surface1,(unsigned char *)surface2, width, height, pitch, 
                                       Amp,a,b,x0,y0,yref, RgbMask );

}


// Texture reference for 2D float texture
texture< uchar4, 2, hipReadModeNormalizedFloat /*hipReadModeElementType*/> tex;

__global__ void Kernel_CartToPol2( unsigned char *surface2, int width, int height, size_t pitch )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel2;

        
    if (x >= width || y >= height) return;
    
    float u = x / (float) width;
    float v = y / (float) height;

    
    float tu = 2* sqrtf( powf(u-0.5,2) + powf(v-0.5,2) );  //R
    float tv = (atan2f( v-0.5 ,  u-0.5) +PI)/(2*PI);       //theta  
          

    pixel2 = (unsigned char *)(surface2 + y*pitch) + 4*x;
        
    pixel2[1] = 255.0* tex2D(tex, tu, tv).y; 
   
}





 int CartToPolK2(hipArray *cuArray ,void *surface2, int width, int height, size_t pitch)
{
       
        // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates
       
    int error =0;  
    int res;
   
    hipChannelFormatDesc channelDesc;
    res= hipGetChannelDesc(&channelDesc, cuArray);
    if  ( res!=0 && error==0) error=1;

    res= hipBindTextureToArray( tex, cuArray, channelDesc);
    if  ( res!=0 && error==0) error=2;
    
    dim3 Db = dim3(16, 16);                   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_CartToPol2<<<Dg,Db>>>( (unsigned char *)surface2, width, height, pitch );

    
    res= hipUnbindTexture(tex);
    if  ( res!=0 && error==0) error=3;
      
    //error =  channelDesc.x + channelDesc.y*100+ + channelDesc.z*10000+ channelDesc.w*1000000  ;
    return error;
    //error = hipGetLastError();
 }
   

  



__global__ void Kernel_Interp(unsigned char *surface2, int width, int height,size_t pitch, float *tbX, float *tbY )
{
        
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    
    unsigned char *pixel2;
        
    if (i >= width || j >= height) return;

   
    float xt = tbX[j*width+i];
    float yt = tbY[j*width+i];
    
    float tu =  xt/(float) width ;    // conversion en coordonnées réduites bitmap
    float tv =  yt/(float) height ;

    
    pixel2 = (unsigned char *)(surface2 + j*pitch) + 4*i;       
    pixel2[1] =  255.0* tex2D(tex, tu, tv).y; 
    
   
}

int InterpK2(hipArray *cuArray ,void *surface2, int width, int height, size_t pitch, float* matX, float* matY)
{
       
        // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates
       
    int error =0;  
    int res;
   
    hipChannelFormatDesc channelDesc;
    res= hipGetChannelDesc(&channelDesc, cuArray);
    if  ( res!=0 && error==0) error=1;

    res= hipBindTextureToArray( tex, cuArray, channelDesc);
    if  ( res!=0 && error==0) error=2;
    
    dim3 Db = dim3(16, 16);                   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_Interp<<<Dg,Db>>>((unsigned char*) surface2,width, height,pitch, matX, matY);
    //Kernel_CartToPol2<<<Dg,Db>>>( (unsigned char *)surface2, width, height, pitch );
    
    res= hipUnbindTexture(tex);
    if  ( res!=0 && error==0) error=3;
      
    //error =  channelDesc.x + channelDesc.y*100+ + channelDesc.z*10000+ channelDesc.w*1000000  ;
    return error;
    //error = hipGetLastError();

}  


/* Copie d'une surface sur une autre

  Les opérations sont effectuées dans cet ordre:
    - on fait tourner la source autour de (xcSrc, ycSrc) d'un angle theta
    - on effectue un scaling (1/ax,1/ay)
    - on place le centre de la figure obtenue en (x0,y0)

  Le calcul fait les opérations à l'envers: connaissant le point de destination M(idest,jdest), il
  faut trouver le point de la source:

     - on calcule les coo de M par rapport à (xcdest,ycdest), puis par rapport à (x0,y0)
     - on effectue une rotation de -theta
     - puis un scaling (ax,ay)
     - puis on calcule les coo de M par rapport au coin du rectangle source

  
  Pas d'utilisation de tex2D
    Mode 1: simple copie (?)   , il est intéressant d'avoir une interp bilinéaire
    Mode 2: les pixels contiennent un index (1,2,3)
            on remplace l'index par Lum[index] ou Alpha[index]
            l'interp bilinéaire a peu d'intérêt

*/


__global__ void KDispSurfaceOnSurface(unsigned char *surf1, int pitch1, int Nx1, int Ny1, 
                                     unsigned char *surf2, int pitch2, int Nx2, int Ny2, 
                                     float x0, float y0,float theta, float ax, float ay,
                                     float xcSrc, float ycSrc, float xcDest, float ycDest,
                                     int AlphaMode, int LumMode,  Tint4 Alpha, Tint4 Lum,
                                     int Mask)
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (idest >= Nx2 || jdest >= Ny2) return;

    pixel2 = (unsigned char *)(surf2 + jdest*pitch2) + 4*idest;
        
    float x = (idest-xcDest) - x0;                      // coo par rapport à x0,y0
    float y = (jdest-ycDest) - y0;                      // x0 et y0 sont exprimés relativement au centre de la destination

    float xp = (x*cos(theta)-y*sin(theta)) * ax;        // rotation  -theta
    float yp = (x*sin(theta)+y*cos(theta)) * ay;        // et scaling (ax,ay)

    float xp0 = xcSrc + xp;                             // coo dans le rectangle source
    float yp0 = ycSrc + yp;


    int x1 = floorf(xp0) ;
    int y1 = floorf(yp0) ;
            
    if ( (x1<0) || (x1>=Nx1) || (y1<0) || (y1>=Ny1)) return;  // ajouter une valeur par défaut ?

    int xp1;
    int yp1;
    if (x1<Nx1-1) xp1= x1+1; else xp1= x1;
    if (y1<Ny1-1) yp1= y1+1; else yp1= y1;
        
    float dx =  xp0-x1;
    float dy =  yp0-y1; 


    for (int i=0; i<4; i++)
      {
        int z1 =   *((unsigned char *)(surf1 + y1*pitch1  + 4*x1+i));
        int z2 =   *((unsigned char *)(surf1 + yp1*pitch1 + 4*x1+i));
        int z3 =   *((unsigned char *)(surf1 + yp1*pitch1 + 4*xp1+i));
        int z4 =   *((unsigned char *)(surf1 + y1*pitch1  + 4*xp1+i));

        float zp;
        float zq;
        float z1a;
        float z2a;
        float z3a;
        float z4a;

        if ((i<3) && (Mask & (1<<i))) 
        {
          switch (LumMode)
          { 
            case 1:   float zp = z1+ dy*(z2-z1);
                      float zq = z4+ dy*(z3-z4);
                      pixel2[i] = (int) (zp+ dx*(zq-zp)+0.499999);                                   
                      break;

            case 2:   if ((z1>=1) && (z1<=3))
                      {
                        z1a = Lum.w[z1-1]; 
                        if (z1a>=0)
                        {
                          pixel2[i] = z1a;
                          /*
                          if ((z2>=1) && (z2<=3)) z2a = Lum[z2-1]; else z2a = z1a;
                          if ((z3>=1) && (z3<=3)) z3a = Lum[z3-1]; else z3a = z1a;
                          if ((z4>=1) && (z4<=3)) z4a = Lum[z4-1]; else z4a = z1a;

                          float zp = z1a+ dy*(z2a-z1a);
                          float zq = z4a+ dy*(z3a-z4a);
                          pixel2[i] = (int) (zp+ dx*(zq-zp)+0.499999);                                   
                          */
                        }
                      }
                      break;
                  
          } 
        }
        else
        if (i==3)
        {
          switch (AlphaMode)
          {
            case 1:   float zp = z1+ dy*(z2-z1);
                      float zq = z4+ dy*(z3-z4);
                      pixel2[i] = (int) (zp+ dx*(zq-zp)+0.499999);                        
                      break;

            case 2:   if ((z1>=1) && (z1<=3))
                      {
                        z1a = Alpha.w[z1-1];                                               
                        if (z1a>=0)
                        {
                          pixel2[i] = z1a;
                          /*
                          if ((z2>=1) && (z2<=3)) z2a = Alpha[z2-1]; else z2a = z1a;
                          if ((z3>=1) && (z3<=3)) z3a = Alpha[z3-1]; else z3a = z1a;
                          if ((z4>=1) && (z4<=3)) z4a = Alpha[z4-1]; else z4a = z1a;

                          float zp = z1a+ dy*(z2a-z1a);
                          float zq = z4a+ dy*(z3a-z4a);
                          pixel2[i] = (int) (zp+ dx*(zq-zp)+0.499999);                                   
                          */
                        }
                      }
                      break;
          }
        
        } 
      } 
}

void DispSurfaceOnSurface(void *surf1, int pitch1, int Nx1, int Ny1, 
                          void *surf2, int pitch2, int Nx2, int Ny2, 
                          float x0, float y0,float theta, float ax, float ay,
                          float xcSrc, float ycSrc, float xcDest, float ycDest,
                          int AlphaMode, int LumMode, Tint4 Alpha, Tint4 Lum, 
                          int Mask, hipStream_t stream)                          
{
  dim3 Db =  dim3(MaxThreadsX, MaxThreadsY);                 
  dim3 Dg = dim3((Nx2+Db.x-1)/Db.x, (Ny2+Db.y-1)/Db.y);

  KDispSurfaceOnSurface<<<Dg,Db,0,stream>>>((unsigned char*)surf1, pitch1, Nx1, Ny1, 
                        (unsigned char*)surf2, pitch2, Nx2, Ny2, 
                        x0, y0,theta, ax, ay, 
                        xcSrc, ycSrc, xcDest, ycDest,
                        AlphaMode, LumMode, Alpha, Lum, Mask );

}


/* Version de DispSurfaceOnSurface avec texture fetching

   On a forcément LumMode=2 ou 0 et AlphaMode=2 ou 0
   Pas de filtrage bilinéaire

*/

// Texture reference for 2D uchar4 texture
texture< uchar4, 2, hipReadModeElementType > tex1;

__global__ void KDispTexOnSurface1(   int Nx1, int Ny1, 
                                     unsigned char *surf2, int pitch2, int Nx2, int Ny2, 
                                     float x0, float y0,float theta, float ax, float ay,
                                     float xcSrc, float ycSrc, float xcDest, float ycDest,
                                     int AlphaMode, int LumMode,  Tint4 Alpha, Tint4 Lum,
                                     int Mask)
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;
        
    unsigned char *pixel2;
        
    if (idest >= Nx2 || jdest >= Ny2) return;

    pixel2 = (unsigned char *)(surf2 + jdest*pitch2) + 4*idest;
        
    float x = (idest-xcDest) - x0;
    float y = (jdest-ycDest) - y0;

    float xp = (x*cos(theta)-y*sin(theta)) * ax;
    float yp = (x*sin(theta)+y*cos(theta)) * ay;

    float xp0 = xcSrc + xp;
    float yp0 = ycSrc + yp;


    int x1 = floorf(xp0) ;
    int y1 = floorf(yp0) ;
            
    if ( (x1<0) || (x1>=Nx1) || (y1<0) || (y1>=Ny1)) return;  // ajouter une valeur par défaut ?

    uchar4 pix = tex2D(tex1,x1,y1);

    int z1a;

    if (LumMode==2)
    {
    if ((pix.x>=1) && (pix.x<=3) && (Mask & 1) )
      {
        z1a = Lum.w[pix.x-1]; 
        if (z1a>=0) pixel2[0] = z1a;
      }
    if ((pix.y>=1) && (pix.y<=3) && (Mask & 2) )
      {
        z1a = Lum.w[pix.y-1]; 
        if (z1a>=0) pixel2[1] = z1a;
      }  
    if ((pix.z>=1) && (pix.z<=3) && (Mask & 4) )
      {
        z1a = Lum.w[pix.z-1]; 
        if (z1a>=0) pixel2[2] = z1a;
      }
    }

    if ((pix.w>=1) && (pix.w<=3) && (AlphaMode==2) )
    {
      z1a = Alpha.w[pix.w-1];                                               
      if (z1a>=0) pixel2[3] = z1a;
    } 
    else
    if ((AlphaMode==3) && (pixel2[3]==Alpha.w[0]))
    {
      pixel2[3]= pix.w*(255.0-Alpha.w[0])/255.0 + Alpha.w[0];
    }
   
}


void DispTexOnSurface(hipArray *SrcArray , int Nx1, int Ny1, 
                          void *surf2, int pitch2, int Nx2, int Ny2, 
                          float x0, float y0,float theta, float ax, float ay,
                          float xcSrc, float ycSrc, float xcDest, float ycDest,
                          int AlphaMode, int LumMode, Tint4 Alpha, Tint4 Lum, 
                          int Mask, hipStream_t stream)                          
{
 
        // Set texture parameters
    tex1.addressMode[0] = hipAddressModeBorder;
    tex1.addressMode[1] = hipAddressModeBorder;
    tex1.filterMode = hipFilterModePoint;
    tex1.normalized = false;    
       
    int error =0;  
    int res;
   
    hipChannelFormatDesc channelDesc;
    res= hipGetChannelDesc(&channelDesc, SrcArray);
    if  ( res!=0 && error==0) error=1;

    res= hipBindTextureToArray( tex1, SrcArray, channelDesc);
    if  ( res!=0 && error==0) error=2;

    dim3 Db =  dim3(MaxThreadsX, MaxThreadsY);                 
    dim3 Dg = dim3((Nx2+Db.x-1)/Db.x, (Ny2+Db.y-1)/Db.y);

    KDispTexOnSurface1<<<Dg,Db, 0,stream>>>( Nx1, Ny1, 
                        (unsigned char*)surf2, pitch2, Nx2, Ny2, 
                        x0, y0,theta, ax, ay, 
                        xcSrc, ycSrc, xcDest, ycDest,
                        AlphaMode, LumMode, Alpha, Lum, Mask );

    res= hipUnbindTexture(tex1);
    if  ( res!=0 && error==0) error=3;
    
}


/*     SMOOTH
          
      On applique un filtre de smooth uniforme NxN à une texture
      LumMode<>0 : on applique le fitre à la luminance sinon on ne fait rien
      AlphaMode<>0 : on applique le fitre à la composante Alpha sinon on ne fait rien

      La première version KSmoothSurface est très mauvaise (peu efficace)
 
      La seconde applique successivement deux filtres 1D (SmoothCol et SmoothRow) et est nettement plus rapide

      TODO: ajouter Mask
*/

__global__ void KSmoothSurface(unsigned char *surf1,unsigned char *surf2, int pitch, int Nx, int Ny, 
                               int N, int AlphaMode, int LumMode )
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (idest >= Nx || jdest >= Ny) return;

    pixel2 =  (unsigned char  *)(surf2 + jdest*pitch) + 4*idest;

    float ss;
    
    int imin =idest-N; if (imin<0) imin=0;
    int imax =idest+N; if (imax>Nx-1) imax=Nx-1;
    int jmin =jdest-N; if (jmin<0) jmin=0;
    int jmax =jdest+N; if (jmax>Ny-1) jmax=Ny-1;
    int Nt=(imax-imin+1)*(jmax-jmin+1);
    //if (Nt=0) return;
    
    if (LumMode)
    {
      for (int k=0; k<3; k++)
      {         
        ss = 0;
        for (int i=imin; i<=imax; i++)
        for (int j=jmin; j<=jmax; j++)
          ss = ss + *((unsigned char *)(surf1 + j*pitch  + 4*i+k));         
        pixel2[k] = ss/Nt;
        
        // pixel2[k] = *((unsigned char *)(surf1 + jdest*pitch  + 4*idest+k));         

      }
    }
    
    if (AlphaMode)
    { ss=0;
      for (int i=imin; i<=imax; i++)
      for (int j=jmin; j<=jmax; j++)      
        ss = ss + *((unsigned char *)(surf1 + j*pitch  + 4*i+3));          

      pixel2[3] = ss/Nt;
    }
    

}

void SmoothSurf(void *surf1, void *surf2, int pitch, int Nx, int Ny, 
                               int N, int AlphaMode, int LumMode )
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsY);    
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);

   KSmoothSurface<<<Dg,Db>>>( (unsigned char*)surf1, (unsigned char*)surf2, pitch, Nx, Ny, N, AlphaMode, LumMode );


}

// Smooth colonne

__global__ void KSmoothSurfaceCol(unsigned char *surf1,unsigned char *surf2, int pitch, int Nx, int Ny,
                                  int N1, int N2, int x0, int y0, int dmax)
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (idest >= Nx || jdest >= Ny) return;

    pixel2 =  (unsigned char  *)(surf2 + jdest*pitch) + 4*idest;

    float ss;
    
    
    int jmin;
    int jmax;
    int Nt;          
    float Kr = 1.0;

    if (dmax>0)
    {
      int d = sqrt(1.0*(idest-x0)*(idest-x0)+1.0*(jdest-y0)*(jdest-y0));
      if (d<=dmax) 
      { 
        Kr = (1.0*d)/dmax; 
        N1 = N1*Kr;
        N2 = N2*Kr;
      }
    }
    
    jmin =jdest-N1; if (jmin<0) jmin=0;
    jmax =jdest+N1; if (jmax>Ny-1) jmax=Ny-1;
    Nt=jmax-jmin+1;

    for (int k=0; k<3; k++)
    {         
      ss = 0;    
      for (int j=jmin; j<=jmax; j++)
        ss = ss + *((unsigned char *)(surf1 + j*pitch  + 4*idest+k));               
      pixel2[k] = ss/Nt;
    }
      
    jmin =jdest-N2; if (jmin<0) jmin=0;
    jmax =jdest+N2; if (jmax>Ny-1) jmax=Ny-1;
    Nt=jmax-jmin+1;
      
    ss=0;      
    for (int j=jmin; j<=jmax; j++)      
      ss = ss + *((unsigned char *)(surf1 + j*pitch  + 4*idest+3));          

    pixel2[3] = ss/Nt;    
    
}


// Smooth Colonne mais sur une texRef

__global__ void KSmoothTexCol(unsigned char *surf2, int pitch, int Nx, int Ny, int N1, int N2, int x0, int y0, int dmax)
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (idest >= Nx || jdest >= Ny) return;

    pixel2 =  (unsigned char  *)(surf2 + jdest*pitch) + 4*idest;
       
    
    int jmin;
    int jmax;
    int Nt;                   
    float Kr = 1.0;

    if (dmax>0)
    {
      int d = sqrt(1.0*(idest-x0)*(idest-x0)+1.0*(jdest-y0)*(jdest-y0));
      if (d<=dmax) 
      { 
        Kr = (1.0*d)/dmax;
        N1 = N1*Kr;
        N2 = N2*Kr;
      }
    }
    jmin =jdest-N1; if (jmin<0) jmin=0;
    jmax =jdest+N1; if (jmax>Ny-1) jmax=Ny-1;
    Nt=jmax-jmin+1;

    int ss[4];
    uchar4 pix;  

	for (int j=0; j<4; j++) { ss[j] = 0; }

    for (int j=jmin; j<=jmax; j++)
    {
      pix = tex2D(tex1,idest,j);         
      
      ss[0] = ss[0]+pix.x;
      ss[1] = ss[1]+pix.y;
      ss[2] = ss[2]+pix.z;
    }     
    for (int i=0;i<3;i++) pixel2[i] = ss[i]/Nt;
      
    jmin =jdest-N2; if (jmin<0) jmin=0;
    jmax =jdest+N2; if (jmax>Ny-1) jmax=Ny-1;
    Nt=jmax-jmin+1;
      
    int s=0;      
    for (int j=jmin; j<=jmax; j++)      
    {
      pix = tex2D(tex1,idest,j);    
      s = s + pix.w;          
    }
    pixel2[3] = s/Nt;    
    
}


// Smooth Row
__global__ void KSmoothSurfaceRow(unsigned char *surf1,unsigned char *surf2, int pitch, int Nx, int Ny, 
                               int N1, int N2, int x0, int y0, int dmax, int dmax2, int ref1, int ref2, int ref3 )
{
    int idest = blockIdx.x*blockDim.x + threadIdx.x;
    int jdest = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
    unsigned char *pixel2;
        
    if (idest >= Nx || jdest >= Ny) return;

    pixel2 =  (unsigned char  *)(surf2 + jdest*pitch) + 4*idest;

    float ss;
    float Kr = 1.0;
    float Kr2 = 1.0;

    //if (Nt=0) return;
    if (dmax>0)
    {
      int d = sqrt(1.0*(idest-x0)*(idest-x0)+1.0*(jdest-y0)*(jdest-y0));
      if (d<=dmax) 
      { 
        Kr = (1.0*d)/dmax;
        N1 = N1*Kr;
        N2 = N2*Kr;
      }
      if (d<=dmax2) 
      { 
        Kr2 = (1.0*d)/dmax2;
      }
    }

    int imin =idest-N1; if (imin<0) imin=0;
    int imax =idest+N1; if (imax>Nx-1) imax=Nx-1;     
    int Nt= imax-imin+1;

    int ref[3];
    ref[0] = ref1;
    ref[1] = ref2;
    ref[2] = ref3;

    for (int k=0; k<3; k++)
    {         
      ss = 0;
      for (int i=imin; i<=imax; i++)        
        ss = ss + *((unsigned char *)(surf1 + jdest*pitch  + 4*i+k));         
      
      pixel2[k] = ref[k] + (ss/Nt-ref[k])*Kr2;               
      //pixel2[k] = ss/Nt;               
    }
    
    imin =idest-N2; if (imin<0) imin=0;
    imax =idest+N2; if (imax>Nx-1) imax=Nx-1;     
    Nt= imax-imin+1;

    ss=0;
    for (int i=imin; i<=imax; i++) 
      ss = ss + *((unsigned char *)(surf1 + jdest*pitch  + 4*i+3));          

    pixel2[3] = ss/Nt; // 255+(ss/Nt-255) * Kr ;    
}


void SmoothSurf2(void *surf1, void *surf2, void *surfDum, int pitch, int Nx, int Ny, 
                               int N1, int N2, int x0, int y0, int dmax, int dmax2, int* ref )
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsY);    
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
  
  KSmoothSurfaceCol<<<Dg,Db>>>( (unsigned char*)surf1, (unsigned char*)surfDum, pitch, Nx, Ny, N1, N2, x0, y0, dmax );
  KSmoothSurfaceRow<<<Dg,Db>>>( (unsigned char*)surfDum, (unsigned char*)surf2, pitch, Nx, Ny, N1, N2, x0, y0, dmax,dmax2, ref[0], ref[1], ref[2]);
}


void SmoothTex2(hipArray *SrcArray, void *surf2, void *surfDum, int pitch, int Nx, int Ny, 
                               int N1, int N2, int x0, int y0, int dmax, int dmax2, int* ref,  hipStream_t stream)
{
  
        // Set texture parameters
    tex1.addressMode[0] = hipAddressModeBorder;
    tex1.addressMode[1] = hipAddressModeBorder;
    tex1.filterMode = hipFilterModePoint;
    tex1.normalized = false;    
       
    int error =0;  
    int res;
   
    hipChannelFormatDesc channelDesc;
    res= hipGetChannelDesc(&channelDesc, SrcArray);
    if  ( res!=0 && error==0) error=1;

    res= hipBindTextureToArray( tex1, SrcArray, channelDesc);
    if  ( res!=0 && error==0) error=2;

    dim3 Db =  dim3(MaxThreadsX, MaxThreadsY);                 
    dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);

    KSmoothTexCol<<<Dg,Db,0,stream>>>( (unsigned char*)surfDum, pitch, Nx, Ny, N1, N2, x0, y0, dmax);

    res= hipUnbindTexture(tex1);
    if  ( res!=0 && error==0) error=3;
    
    KSmoothSurfaceRow<<<Dg,Db,0,stream>>>( (unsigned char*)surfDum, (unsigned char*)surf2, pitch, Nx, Ny, N1, N2, x0, y0, dmax,dmax2, ref[0],ref[1],ref[2]);

}

/*                       
*/

__global__ void KtexSumX( int* outD, int NtotX, int NtotY, int NbUx, int NbUy, int ref)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;
  int idy = threadIdx.y;
  int Bidy = blockIdx.y;

  int NthreadX = blockDim.x; 
  int NblockX = gridDim.x;
  int NthreadY = blockDim.y; 
  int NblockY = gridDim.y;                       
    
  __shared__ int A0[4096];     // size = Nthread max 

  int i0 = Bidx*NthreadX*NbUx+ idx*NbUx;
  int j0 = Bidy*NthreadY*NbUy+ idy*NbUy;


  int IA0 = idx + NthreadX*idy; 
  A0[IA0]=0;
  for (int i=0; i<NbUx; i++)
  for (int j=0; j<NbUy; j++)
  { if ((i0+i<NtotX)&& (j0+j<NtotY))
    { uchar4 pix = tex2D(tex1,i0+i,j0+j);
      if (pix.x==ref) A0[IA0]++ ;
    }
  } 

  __syncthreads();
 
  if ((idx==0) && (idy==0))
  {
   outD[Bidx+NblockX*Bidy] =0;
   for (int i=0;i< NthreadX*NthreadY; i++)   
   {outD[Bidx+NblockX*Bidy] += A0[i]; }
  }

}

__global__ void KtexSumY( int* outD, int NtotX, int NtotY, int NbUx, int NbUy, int ref)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;
  int idy = threadIdx.y;
  int Bidy = blockIdx.y;

  int NthreadX = blockDim.x; 
  int NblockX = gridDim.x;
  int NthreadY = blockDim.y; 
  int NblockY = gridDim.y;                       
    
  __shared__ int A0[4096];     // size = Nthread max 

  int i0 = Bidx*NthreadX*NbUx+ idx*NbUx;
  int j0 = Bidy*NthreadY*NbUy+ idy*NbUy;


  int IA0 = idx + NthreadX*idy; 
  A0[IA0]=0;
  for (int i=0; i<NbUx; i++)
  for (int j=0; j<NbUy; j++)
  { if ((i0+i<NtotX)&& (j0+j<NtotY))
    { uchar4 pix = tex2D(tex1,i0+i,j0+j);
      if (pix.y==ref) A0[IA0]++ ;
    }
  } 

  __syncthreads();
 
  if ((idx==0) && (idy==0))
  {
   outD[Bidx+NblockX*Bidy] =0;
   for (int i=0;i< NthreadX*NthreadY; i++)   
   {outD[Bidx+NblockX*Bidy] += A0[i]; }
  }

}


__global__ void KtexSumZ( int* outD, int NtotX, int NtotY, int NbUx, int NbUy, int ref)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;
  int idy = threadIdx.y;
  int Bidy = blockIdx.y;

  int NthreadX = blockDim.x; 
  int NblockX = gridDim.x;
  int NthreadY = blockDim.y; 
  int NblockY = gridDim.y;                       
    
  __shared__ int A0[4096];     // size = Nthread max 

  int i0 = Bidx*NthreadX*NbUx+ idx*NbUx;
  int j0 = Bidy*NthreadY*NbUy+ idy*NbUy;


  int IA0 = idx + NthreadX*idy; 
  A0[IA0]=0;
  for (int i=0; i<NbUx; i++)
  for (int j=0; j<NbUy; j++)
  { if ((i0+i<NtotX)&& (j0+j<NtotY))
    { uchar4 pix = tex2D(tex1,i0+i,j0+j);
      if (pix.z==ref) A0[IA0]++ ;
    }
  } 

  __syncthreads();
 
  if ((idx==0) && (idy==0))
  {
   outD[Bidx+NblockX*Bidy] =0;
   for (int i=0;i< NthreadX*NthreadY; i++)   
   {outD[Bidx+NblockX*Bidy] += A0[i]; }
  }

}

__global__ void KtexSumW( int* outD, int NtotX, int NtotY, int NbUx, int NbUy, int ref)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;
  int idy = threadIdx.y;
  int Bidy = blockIdx.y;

  int NthreadX = blockDim.x; 
  int NblockX = gridDim.x;
  int NthreadY = blockDim.y; 
  int NblockY = gridDim.y;                       
    
  __shared__ int A0[4096];     // size = Nthread max 

  int i0 = Bidx*NthreadX*NbUx+ idx*NbUx;
  int j0 = Bidy*NthreadY*NbUy+ idy*NbUy;


  int IA0 = idx + NthreadX*idy; 
  A0[IA0]=0;
  for (int i=0; i<NbUx; i++)
  for (int j=0; j<NbUy; j++)
  { if ((i0+i<NtotX)&& (j0+j<NtotY))
    { uchar4 pix = tex2D(tex1,i0+i,j0+j);
      if (pix.w==ref) A0[IA0]++ ;
    }
  } 

  __syncthreads();
 
  if ((idx==0) && (idy==0))
  {
   outD[Bidx+NblockX*Bidy] =0;
   for (int i=0;i< NthreadX*NthreadY; i++)   
   {outD[Bidx+NblockX*Bidy] += A0[i]; }
  }

}


int TexSum(hipArray *SrcArray , int NtotX, int NtotY, int* Odata, int Comp, int ref )
{ 
  int tbres[2048];
  int res;
  int* Odata1;

  int NthreadX =  MaxThreadsX;
  int NblockX =1024;

  while ((NthreadX*NblockX>NtotX)&&(NblockX>1)) NblockX = NblockX/2;  
  int NbUx = NtotX/(NthreadX*NblockX);
  if (NtotX % (NthreadX*NblockX) !=0) {NbUx++;}

  
  while ((NblockX>NbUx)&&(NblockX>1)) {
    NblockX = NblockX/2;  
    NbUx = NbUx*2;
  }

  int NthreadY =  MaxThreadsY;
  int NblockY =1024;

  while ((NthreadY*NblockY>NtotY)&&(NblockY>1)) NblockY = NblockY/2;  
  int NbUy = NtotY/(NthreadY*NblockY);
  if (NtotY % (NthreadY*NblockY) !=0) {NbUy++;}
  
  while ((NblockY>NbUy)&&(NblockY>1)) {
    NblockY = NblockY/2;  
    NbUy = NbUy*2;
  }
    
  if (Odata !=NULL) Odata1=Odata; else hipMalloc((void**) &Odata1, NblockX*NblockY* sizeof(int));        

  tex1.addressMode[0] = hipAddressModeBorder;
  tex1.addressMode[1] = hipAddressModeBorder;
  tex1.filterMode = hipFilterModePoint;
  tex1.normalized = false;    
       
  int error =0;  
 
   
  hipChannelFormatDesc channelDesc;
  res= hipGetChannelDesc(&channelDesc, SrcArray);
  if  ( res!=0 && error==0) error=1;

  res= hipBindTextureToArray( tex1, SrcArray, channelDesc);
  if  ( res!=0 && error==0) error=2;

  if (error!=0)
	  {
		  if (Odata == NULL) hipFree(Odata1);
		  return -error;

	  }


  dim3 Db = dim3(NthreadX, NthreadY);    
  dim3 Dg = dim3(NblockX, NblockY);    

  
  switch (Comp)
  {
    case 1: KtexSumX<<<Dg,Db>>>(Odata1,NtotX,NtotY,NbUx,NbUy, ref ); 
            break;
    case 2: KtexSumY<<<Dg,Db>>>(Odata1,NtotX,NtotY,NbUx,NbUy, ref ); 
            break;
    case 3: KtexSumZ<<<Dg,Db>>>(Odata1,NtotX,NtotY,NbUx,NbUy, ref ); 
            break;
    case 4: KtexSumW<<<Dg,Db>>>(Odata1,NtotX,NtotY,NbUx,NbUy, ref ); 
            break;
  }

  res= hipUnbindTexture(tex1);

  hipMemcpy(tbres,Odata1,NblockX*NblockY*sizeof(int),hipMemcpyDeviceToHost);
  
  if (Odata == NULL) hipFree(Odata1);

  res = 0;
  for (int i=0;i<NblockX*NblockY;i++) res+=tbres[i];
  return res;

}


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié
__global__ void 
FillTexByte(void *surface, int width, int height, size_t pitch,unsigned char* src, int Mask, float Ascale, float Bscale)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
        
    if (x >= width || y >= height) return;        
    
	//Ascale = 1.0;
	//Bscale = 0;


	int w = src[x + width*y];
    w =  Ascale*w + Bscale;

	if (w<0)   { w = 0;}
	else
	if (w>253) { w = 253;}

	pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;
        
	pixel1[3] =255;
    // alpha n'est pas modifié sauf s'il fait partie du masque
    for (int i=0;i<4;i++)    
    {  if (Mask & (1<<i))  pixel1[i] = w;  } 
}

int FillByteTexture(void* LinearMem, int Nx, int Ny, size_t PitchMem, unsigned char* Image, int ColorMask, float Ascale, float Bscale)
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                   
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
    
  FillTexByte<<<Dg,Db>>>(LinearMem,Nx,Ny,PitchMem,Image,ColorMask, Ascale, Bscale);
  return 0;
}
