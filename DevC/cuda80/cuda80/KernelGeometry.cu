
#include <hip/hip_runtime.h>
extern int MaxThreadsPerBlock;
extern int MaxThreadsX;
extern int MaxThreadsY;


__global__ void Kernel_Rings1(unsigned char *surface1, int width, int height, size_t pitch,
                                       float Amp, float a, float b, float Rt, int x0, int y0, float yref, int Mask )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
           
    if (x >= width || y >= height) return;

    pixel1 = (unsigned char *)(surface1 + y*pitch) + 4*x;
       
    float R =  sqrtf( powf(x-x0,2) + powf(y-y0,2) ) ;
    float ZR  = Amp*sin(a*R+b);
    if (Rt>0) ZR = ZR*expf(-R/Rt);
       
	int w = yref + ZR;     

    if (w<0) w=0;
    else
    if (w>253) w=253;

	for (int i=0;i<3;i++)
    {
      if (Mask & (1<<i))  pixel1[i] = w;  
    }
    pixel1[3] =w; // alpha comme les autres 
}

void BuildRingsK1( void *surface1, int width, int height, size_t pitch, 
                      float Amp, float a, float b, float Rt, int x0, int y0, float yref, int RgbMask, int mode )
{
    dim3 Db = dim3(MaxThreadsX, MaxThreadsY);                  
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_Rings1<<<Dg,Db>>>((unsigned char *)surface1, width, height, pitch, Amp,a,b,Rt,x0,y0,yref, RgbMask );

}
