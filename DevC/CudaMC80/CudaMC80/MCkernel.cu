#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <math.h>
#include "MC2.h"

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur r�el en vecteur complexe
__global__ void
RealToCpx(const double *A, Complex *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        B[i].x = A[i];
        B[i].y = 0;
    }    
}

// Conversion d'un vecteur complexe en vecteur r�el
__global__ void
CpxToReal(const Complex *A, double *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        B[i] = A[i].x;        
    }
}


// Multiplie point par point un vecteur complex par un vecteur r�el 
__global__ void
MulCpx( Complex *A, const double *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i].x = A[i].x*B[i];
        A[i].y = A[i].y*B[i];
    }    
}

// Applique y = at*x +bt � chaque point d'un vecteur r�el 
__global__ void
LinearTransform(double *A, int numElements, double at, double bt)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = A[i]* at + bt;        
    }    
}




// Remplissage de la linearmem (tableau de pixels) associ�e � la texture avec le tableau de r�el
// Alpha n'est pas modifi�
__global__ void 
FillTex(void *surface, int width, int height, size_t pitch, double* src, int Mask)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
        
    if (x >= width || y >= height) return;        
    
    double w = src[x + width*y];
    
    if (w<0) {w=0;}
    if (w>253) {w=253;}
                      
    
    pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;
        
    //pixel1[3] = 255;                     // alpha = 255 sauf s'il fait partie du masque
    for (int i=0;i<4;i++)    
    {  if (Mask & (1<<i))  pixel1[i] = w;  } 
}

// Remplissage de la linearmem (tableau de pixels) associ�e � la texture avec le tableau de bytes
// Alpha n'est pas modifi�
__global__ void 
FillTexByte(void *surface, int width, int height, size_t pitch, char* src, int Mask)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
        
    if (x >= width || y >= height) return;        
    
    char w = src[x + width*y];                      
    
    pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;
        
    // alpha n'est pas modifi� sauf s'il fait partie du masque
    for (int i=0;i<4;i++)    
    {  if (Mask & (1<<i))  pixel1[i] = w;  } 
}


// Remplissage de la linearmem (tableau de pixels) associ�e � la texture avec le tableau de r�el
// Alpha autorise l'affichage au dessus d'un certain seuil
__global__ void 
FillTexTh(void *surface, int width, int height, size_t pitch, double* src, int Mask, int th, int pixValue)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
        
    if (x >= width || y >= height) return;        
    
    double w = src[x + width*y];
    
    if (w<0) {w=0;}
    if (w>253) {w=253;}
                      
    
    pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;
        
    if (pixel1[3]>=th) 
    for (int i=0;i<3;i++)    
    {  if (Mask & (1<<i))  pixel1[i] = w;  } 
    else
    for (int i=0;i<3;i++)    
    {  if (Mask & (1<<i))  pixel1[i] = pixValue >> (i*8);  } 
    
    
}


// Processus auto-r�gressif X2 = a*X1 + b*X0 + N0;
 
__global__ void 
AutoRegK(double* X0, double* X1, double* X2, double* N0, int numElements, double a, double b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {   
      X2[i] = a*X1[i] + b*X0[i] + N0[i];
    }
}


// Expansion 
// On applique une interpolation bi-lin�aire � la source
__global__ void Kernel_Expansion1(double *tb1, double *tb2, int width, int height, double Dx, double x0, double Dy, double y0  )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    
        
    if (x >= width || y >= height) return;

    
    double xt = (x-x0)/Dx +x0;
    double yt = (y-y0)/Dy +y0;
   
    int x1 = ((int) xt) % width ;
    int y1 = ((int) yt) % height;

    int xp1 = (x1+1) % width;
    int yp1 = (y1+1) % height;
    
    double z1 =   tb1[width*y1+x1];
    double z2 =   tb1[width*yp1+x1];
    double z3 =   tb1[width*yp1+xp1];
    double z4 =   tb1[width*y1+xp1];

    double dx =  xt-floorf(xt);
    double dy =  yt-floorf(yt); 

    double zp = z1+ dy*(z2-z1);
    double zq = z4+ dy*(z3-z4);
    double ZR = zp+ dx*(zq-zp);
            
    tb2[width*y+x] = ZR;    
   
}

// Transformation Cartesian To Polar
// On applique une interpolation bi-lin�aire � la source
__global__ void Kernel_CartToPol1(double *tb1, double *tb2, int width, int height )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
        
    if (x >= width || y >= height) return;        

    double R = 2* sqrtf( powf(x-width/2,2) + powf(y-height/2,2) );
    double theta = (atan2f( y-height/2 ,  x-width/2) +PI)*height/(2*PI);
  
    int x1 = ((int) R) % width ;
    int y1 = ((int) theta) % height;

    int xp1 = (x1+1) % width;
    int yp1 = (y1+1) % height;

    double z1 =   tb1[width*y1+x1];
    double z2 =   tb1[width*yp1+x1];
    double z3 =   tb1[width*yp1+xp1];
    double z4 =   tb1[width*y1+xp1];

    double dx =  theta-floorf(theta);
    double dy =  R-floorf(R); 

    double zp = 1.0*z1+ dy*(1.0*z2-z1);
    double zq = 1.0*z4+ dy*(1.0*z3-z4);
    double ZR = zp+ dx*(zq-zp);
  
    tb2[width*y+x] = ZR;  
   
}



int TMCstruct :: ComputeNoise(double* noise1, double* filter1, hiprandGenerator_t Agenerator, hipStream_t stream )
{   

   //g�n�rer noise1 
   hiprandStatus_t res; 

   hiprandSetStream(Agenerator,stream);
   res = hiprandGenerateNormalDouble( Agenerator, noise1, Nx * Ny, mu, sigma);    

   //transformer noise1 en complexes 
   RealToCpx<<< Nx, Ny, 0, stream>>>(noise1,fftNoise ,Nx * Ny);
   // DFT de noise1==> fftNoise

   hipfftSetStream(fftPlan,stream);
   hipfftExecZ2Z(fftPlan, (Complex *)fftNoise, (Complex *)fftNoise, HIPFFT_FORWARD );

   // multiplier fftNoise par filter                                  
   MulCpx<<<Nx,Ny,0, stream>>>(fftNoise,filter1,Nx*Ny);
  
   // DFT inverse du r�sultat
   hipfftExecZ2Z(fftPlan, (Complex *)fftNoise, (Complex *)fftNoise, HIPFFT_BACKWARD );
    
   // La partie r�elle donne noise
   CpxToReal<<<Nx,Ny,0,stream>>>(fftNoise, noise1,Nx*Ny);

   return 0;
}


int TMCstruct :: Autoreg()
{
  AutoRegK<<<Nx,Ny>>>( Xn0, Xn1, Xn2, Noise, Nx*Ny, aReg, bReg);

  if ( DxF !=1.0f || DyF !=1.0f ) 
  {
    dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
    Kernel_Expansion1<<<Dg,Db>>>(Xn1, Xn0, Nx, Ny, DxF, X0F, DyF, Y0F  );
    Kernel_Expansion1<<<Dg,Db>>>(Xn2, Xn1, Nx, Ny, DxF, X0F, DyF, Y0F  );
  }
  else
  {
    hipMemcpy( Xn0, Xn1, Nx*Ny*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy( Xn1, Xn2, Nx*Ny*sizeof(double),hipMemcpyDeviceToDevice);    
  }
  return 0;
}

                            



extern "C"    {
  int InitK2();
  int DoneK2();
  int UpdateK2();
}


// Texture reference for 2D float texture
texture< uchar4, 2, hipReadModeNormalizedFloat /*hipReadModeElementType*/> tex;

__global__ void Kernel_CartToPol2( unsigned char *surface2, int width, int height, size_t pitch )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel2;

        
    if (x >= width || y >= height) return;
    
    float u = x / (float) width;
    float v = y / (float) height;

    
    float tu = 2* sqrtf( powf(u-0.5,2) + powf(v-0.5,2) );  //R
    float tv = (atan2f( v-0.5 ,  u-0.5) +PI)/(2*PI);       //theta  
          

    pixel2 = (unsigned char *)(surface2 + y*pitch) + 4*x;
        
    pixel2[1] = 255.0* tex2D(tex, tu, tv).y; 
   
}


extern "C"  int CartToPolK2(hipArray *cuArray ,void *surface2, int width, int height, size_t pitch)
{
       
        // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates
       
    int error =0;  
    int res;
   
    hipChannelFormatDesc channelDesc;
    res= hipGetChannelDesc(&channelDesc, cuArray);
    if  ( res!=0 && error==0) error=1;

    res= hipBindTextureToArray( tex, cuArray, channelDesc);
    if  ( res!=0 && error==0) error=2;
    
    dim3 Db = dim3(MaxThreadsX, MaxThreadsX );                   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    Kernel_CartToPol2<<<Dg,Db>>>( (unsigned char *)surface2, width, height, pitch );

    
    res= hipUnbindTexture(tex);
    if  ( res!=0 && error==0) error=3;
      
    //error =  channelDesc.x + channelDesc.y*100+ + channelDesc.z*10000+ channelDesc.w*1000000  ;
    return error;
    //error = hipGetLastError();

   
}




int TMCstruct :: ComputeNoise2( Complex* fftNoise1, double* filter1)
{   
   //g�n�rer noise1 
   mu=0;
   sigma=1;
   hiprandStatus_t res = hiprandGenerateNormalDouble( generator, Noise, Nx*Ny, mu, sigma);
   //transformer Noise en complexes 
   RealToCpx<<<Nx,Ny>>>(Noise, fftNoise1 ,Nx*Ny);
   // DFT de noise1==> fftNoise
   hipfftExecZ2Z(fftPlan, (Complex *)fftNoise1, (Complex *)fftNoise1, HIPFFT_FORWARD );

   // multiplier fftNoise par filter
   MulCpx<<<Nx,Ny>>>(fftNoise1,filter1,Nx*Ny);
  
   
   return 0;
}


// Processus auto-r�gressif Xf2 = a*Xf1 + b*Xf0 + Nf0;
 
__global__ void 
AutoRegK2(Complex* Xf0, Complex* Xf1, Complex* Xf2, Complex* Nf0, int numElements, double* a, double* b, double* c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {   
      Xf2[i].x = a[i]*Xf1[i].x + b[i]*Xf0[i].x + c[i]*Nf0[i].x;
      Xf2[i].y = a[i]*Xf1[i].y + b[i]*Xf0[i].y + c[i]*Nf0[i].y;

    }
}

int TMCstruct :: Autoreg2()
{
  AutoRegK2<<<Nx,Ny>>>(Xnf0, Xnf1, Xnf2, fftNoise, Nx*Ny, afReg, bfReg, cfReg);

  hipMemcpy( Xnf0, Xnf1, Nx*Ny*sizeof(Complex), hipMemcpyDeviceToDevice);
  hipMemcpy( Xnf1, Xnf2, Nx*Ny*sizeof(Complex), hipMemcpyDeviceToDevice);    

  return 0;
}


int TMCstruct :: InitK2() 
{
  hiprandStatus_t curandResult = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed( generator, seed);

  //if (curandResult != HIPRAND_STATUS_SUCCESS)  { }

  hipfftPlan2d(&fftPlan, Nx, Ny, HIPFFT_Z2Z);

  //ComputeNoise2(Xnf1, Filter);
  //ComputeNoise2(Xnf0, Filter);

  for (int i=0;i<500*Nsample;i++)
  {
    ComputeNoise2( fftNoise,Filter);
    Autoreg2();
  }
  return 0;
}

int TMCstruct :: DoneK2()
{
  if (generator!=NULL)
  {
  hiprandDestroyGenerator(generator);
  hipfftDestroy(fftPlan);

  generator = NULL;
  fftPlan = NULL;
  }
  return 0;
}


int TMCstruct :: UpdateK2()
{
  for (int i=0;i< Nsample; i++)  
  {
    ComputeNoise2(fftNoise, Filter);
    Autoreg2();
  }

  hipfftExecZ2Z(fftPlan, Xnf2, Xnf2, HIPFFT_BACKWARD );
    
   // La partie r�elle donne noise
  CpxToReal<<<Nx,Ny>>>(Xnf2, Noise,Nx*Ny);

  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                   
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
  
  // Normaliser la DFT inverse: multiplication par  1.0/(Nx*Ny)  
  // et r�gler la valeur moyenne et l'amplitude
  //LinearTransform<<<Nx,Ny>>>(Noise ,Nx*Ny , 1.0/(Nx*Ny), 0);
  LinearTransform<<<Nx,Ny>>>(Noise ,Nx*Ny , Atransform/(1.0*Nx*Ny), Btransform);
  FillTex<<<Dg,Db>>>(LinearMem,Nx,Ny,PitchMem,Noise,ColorMask);
  return 0;
}



__global__ void Ksum(double* A, double* outD, int Ntot, int NbU)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;

  int Nthread = blockDim.x; 
  int Nblock = gridDim.x;
                          
    
  __shared__ double A0[4096];     // size = Nthread max 


  int i0 = Bidx*Nthread*NbU+ idx*NbU;
  A0[idx]=0;
  for (int i=0; i<NbU;i++) { if (i0+i<Ntot) A0[idx]+= A[i0+i];} 

  __syncthreads();
 
  if (idx==0)
  {
   outD[Bidx] =0;
   for (int i=0;i< Nthread;i++){outD[Bidx] += A0[i]; }
  }

}



double CudaSum(double* A, int Ntot,double* Odata)
{ 
  double tbres[2048];
  double res;
  double* Odata1;

  int Nthread =  MaxThreadsPerBlock;
  int Nblock =1024;

  while ((Nthread*Nblock>Ntot)&&(Nblock>1)) Nblock = Nblock/2;  
  int NbU = Ntot/(Nthread*Nblock);
  if (Ntot % (Nthread*Nblock) !=0) {NbU++;}

  
  while ((Nblock>NbU)&&(Nblock>1)) {
    Nblock = Nblock/2;  
    NbU = NbU*2;
  }
  
  if (Odata !=NULL) Odata1=Odata; else hipMalloc((void**) &Odata1, Nblock* sizeof(double));        
  
  Ksum<<<Nblock,Nthread>>>(A,Odata1,Ntot,NbU);    
  hipMemcpy(tbres,Odata1,Nblock*sizeof(double),hipMemcpyDeviceToHost);
  
  if (Odata == NULL) hipFree(Odata1);

  res = 0;
  for (int i=0;i<Nblock;i++) res+=tbres[i];
  return res;
}

__global__ void KSqrSum(double* A, double* outD, int Ntot, int NbU)
{
  int idx = threadIdx.x;
  int Bidx = blockIdx.x;

  int Nthread = blockDim.x; 
  int Nblock = gridDim.x;
                          
    
  __shared__ double A0[4096];     // size = Nthread max 


  int i0 = Bidx*Nthread*NbU+ idx*NbU;
  A0[idx]=0;
  for (int i=0; i<NbU;i++) { if (i0+i<Ntot) A0[idx]+=  A[i0+i]*A[i0+i];} 

  __syncthreads();
 
  if (idx==0)
  {
   outD[Bidx] =0;
   for (int i=0;i< Nthread;i++){outD[Bidx] += A0[i]; }
  }

}



double CudaSqrSum(double* A, int Ntot,double* Odata)
{ 
  double tbres[2048];
  double res;
  double* Odata1;

  int Nthread =  MaxThreadsPerBlock;
  int Nblock =1024;

  while ((Nthread*Nblock>Ntot)&&(Nblock>1)) Nblock = Nblock/2;  
  int Nb = Ntot/(Nthread*Nblock);
  if (Ntot % (Nthread*Nblock) !=0) {Nb++;}

  
  while ((Nblock>Nb)&&(Nblock>1)) {
    Nblock = Nblock/2;  
    Nb = Nb*2;
  }
  
  if (Odata !=NULL) Odata1=Odata; else hipMalloc((void**) &Odata1, Nblock* sizeof(double));        
  
  KSqrSum<<<Nblock,Nthread>>>(A,Odata1,Ntot,Nb);    
  hipMemcpy(tbres,Odata1,Nblock*sizeof(double),hipMemcpyDeviceToHost);
  
  if (Odata == NULL) hipFree(Odata1);

  res = 0;
  for (int i=0;i<Nblock;i++) res+=tbres[i];
  return res;
}


__global__ void KGaborFilter1(double* filter, double* Vr, int width, int height, double ss , double r0, double sr0, double stheta0 )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
            
    if (i >= width || j >= height) return;

    double x = i;
    double y = j;

    if (i> width/2)  x = width-i;
    if (j> height/2) y = height-j;
    
    #define Eps 1E-6;
    double r = sqrt(x*x+ y*y)+Eps;

    double theta;
    if (x>0) theta= atan2( y, x); else theta = PI/2;

    //double ff =  exp( cos(2*theta)/stheta0 )                                                          
    //             *
    //             exp(-0.5*pow(log(r/r0),2)/log(1+pow(sr0,2))) * pow(r0/r,3)*ss*r;
    
	// Correction Jonathan 7-12-16
	double ff =  exp( cos(2*theta)/(4*pow(stheta0,2) ) )                                                          
                 *
                 exp(-0.5*pow(log(r/r0),2)/log(1+pow(sr0,2))) * pow(r0/r,3)*4*pow(ss*r,3);
                 

    filter[i+j*width] = ff;    
    if (i>0 || j>0) Vr[i+j*width] =  ff/(4*pow(ss*r,3)); else Vr[i+j*width] = 0;
    

}


__global__ void KGaborFilter2(double* filter, int NumElements, double fMul )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i >NumElements) return;

    filter[i] = sqrt(filter[i]*fMul);
}


__global__ void  KparamAR(double* a, double* b, double* c, double ss, double dtAR, int width, int height)
{
    #define eps 1E-12;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
            
    if (i >= width || j >= height) return;
   
    int  x= i;
    int  y= j;

    if (i> width/2)  x = width-i;
    if (j> height/2) y = height-j;
  
    double r = sqrt( (double)x*x + (double)y*y )+Eps;

    a[i+j*width] =  2-dtAR*2*ss*r- pow(dtAR*ss*r,2);
    b[i+j*width] = -1+dtAR*2*ss*r;
    // c[i+j*width] =  50* pow(dtAR,2);
	// Correction Jonathan 7-12-16
	c[i+j*width] =  1;
  
}

void InitConstants();

void TMCstruct :: InstallLogGaborFilterK(double dtAR, double ss , double r0, double sr0, double stheta0)
{  
  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
  KparamAR<<<Dg,Db>>>(afReg,bfReg,cfReg,ss,dtAR,Nx,Ny);

  double fN;
  KGaborFilter1<<<Dg,Db>>>(Filter,Vdum ,Nx,Ny ,ss ,r0, sr0, stheta0);
    
  fN = CudaSum(Vdum,Nx*Ny,NULL);
  
 // KGaborFilter2<<<Nx,Ny>>>(Filter,Nx*Ny, Nx*Ny/dtAR/fN);
  // Correction Jonathan 7-12-16
  KGaborFilter2<<<Nx,Ny>>>(Filter,Nx*Ny, Nx*Ny*pow(dtAR,3)/fN);
} 

__global__ void 
AutoRegPinkK(double* X0, double* X1, double* Y, double* C0, double* D0, double* LastF, int numElts, int Nc)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElts)
    {    
      for (int j=0; j<Nc; j++)
        LastF[j*numElts+i] = (X0[i]+X1[i])*D0[j] - C0[j]*LastF[j*numElts+i];
      
      double w = 0;
      for (int j=0; j<Nc; j++) w = w+ LastF[j*numElts+i];  
      Y[i] = w;     
    }
}
      
int TMCstruct :: AutoregPink(double* Noise1, double* LastF, hipStream_t stream)
{
  AutoRegPinkK<<<(Nx*Ny+MaxThreadsPerBlock-1)/MaxThreadsPerBlock,MaxThreadsPerBlock, 0, stream>>>(Noise, Noise1, Noise2, 
                                                                                       C0, D0, LastF, Nx*Ny, Nc);

  hipMemcpyAsync( Noise1, Noise, Nx*Ny*sizeof(double), hipMemcpyDeviceToDevice, stream); 
  
  return 0;
}

int TMCstruct :: UpdateKpink(hipStream_t stream)
{
  if (RstState) ComputeNoise(Noise, Filter, RstGenerator, stream);  
  else ComputeNoise(Noise, Filter, generator, stream);   

  if (RstState) 
     AutoregPink(Noise1,LastF, stream);
     else AutoregPink(Noise1_cont,LastF_cont, stream);
  
  return 0;
}

int TMCstruct :: UpdateKpink2(hipStream_t stream)
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                  
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
  LinearTransform<<<Nx,Ny,0, stream>>>(Noise2, Nx*Ny , Atransform, Btransform);

  if (AlphaTh>0)  FillTexTh<<<Dg,Db,0, stream>>>(LinearMem,Nx,Ny,PitchMem,Noise2,ColorMask,AlphaTh,AlphaThV );
  else FillTex<<<Dg,Db,0,stream>>>(LinearMem,Nx,Ny,PitchMem,Noise2,ColorMask );
               
  return 0;
}


int TMCstruct :: InitKpink(double* aa, double* bb, int* Nt)
{
  hipStream_t    stream = 0;

  mu = 0;
  sigma = 1;
  hiprandStatus_t curandResult;
  
  hiprandSetStream(generator, stream);
  hiprandSetStream(RstGenerator, stream);

  hiprandSetPseudoRandomGeneratorSeed( generator, seed);
  hiprandSetPseudoRandomGeneratorSeed( RstGenerator, seed);
    
  //hipfftSetStream(fftPlan, stream);
 
  *aa=0;
  *bb=0;
  *Nt=0;
  for (int i=0;i<250;i++)
  {
    ComputeNoise(Noise,Filter, generator,0);
    AutoregPink(Noise1, LastF,0);
    if (i>=50)
    {
    *aa = *aa + CudaSum(Noise2,Nx*Ny,NULL);
    *bb = *bb + CudaSqrSum(Noise2,Nx*Ny,NULL);
    *Nt = *Nt+1;
    }
  }
    
  *Nt = *Nt*Nx*Ny; 

  hiprandSetGeneratorOffset(generator, Nx*Ny*250 );

  hipMemcpy( Noise1_init, Noise1 , Nx*Ny* sizeof(double), hipMemcpyDeviceToDevice);
  hipMemcpy( LastF_init, LastF,    Nx*Ny*Nc* sizeof(double), hipMemcpyDeviceToDevice);
  
  hipMemcpy( Noise1_cont, Noise1 , Nx*Ny* sizeof(double), hipMemcpyDeviceToDevice);
  hipMemcpy( LastF_cont, LastF,    Nx*Ny*Nc* sizeof(double), hipMemcpyDeviceToDevice);
  
  return 0;
}

int TMCstruct :: DoneKpink()
{
  
  return 0;
}


// On remplit tb avec 1 pour les pixels situ�s dans le quadrilat�re Pts et tels que pixel[k]>=th
// k= 0,1,2,3
__global__ void 
KtexFillRect(void* surface, double* tb, int width, int height, size_t pitch, float2* Pts, int k, float th)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned char *pixel1;
        
    if (x >= width || y >= height) return;        
    
    pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;

    if ( 
        ((Pts[1].y-Pts[0].y)*(x-Pts[0].x)-( y-Pts[0].y)*(Pts[1].x-Pts[0].x)>=0)
        &&
        ((Pts[2].y-Pts[1].y)*(x-Pts[1].x)-( y-Pts[1].y)*(Pts[2].x-Pts[1].x)>=0)
        &&
        ((Pts[3].y-Pts[2].y)*(x-Pts[2].x)-( y-Pts[2].y)*(Pts[3].x-Pts[2].x)>=0)
        &&
        ((Pts[0].y-Pts[3].y)*(x-Pts[3].x)-( y-Pts[3].y)*(Pts[0].x-Pts[3].x)>=0)
        &&
        (pixel1[k]>=th)
        ) 
        tb[x + width*y] = 1;
           
        
    
}

void texFillRect(void* surface, double* tb, int width, int height, size_t pitch, float2* Pts, int k, float th)
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                   
  dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

  KtexFillRect<<<Dg,Db>>>( surface, tb, width, height, pitch, Pts, k,  th); 
  
}

int FillByteTexture(void* LinearMem, int Nx, int Ny, size_t PitchMem, char* Image, int ColorMask)
{
  dim3 Db = dim3(MaxThreadsX, MaxThreadsX);                   
  dim3 Dg = dim3((Nx+Db.x-1)/Db.x, (Ny+Db.y-1)/Db.y);
    
  FillTexByte<<<Dg,Db>>>(LinearMem,Nx,Ny,PitchMem,Image,ColorMask);
  return 0;
}
