
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <math.h>

#define PI 3.1415926536f


float*      matX;
float*      matY;

__global__ void Kernel_Interp(float *tb1, float *tb2, float *tbX, float *tbY, int width, int height, float a1,float b1, float c1, float a2,float b2, float c2 )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    
        
    if (i >= width || j >= height) return;

    int i0 = int (a1*i+b1) % 61;       // conversion de (i,j) en cm �cran puis en unit�s tbX/tbY
    int j0 = int (a1*j+c1) % 61 ;


    float xt = tbX[width*j0+i0];     // coordonn�es de la transform�e dans le cortex en mm
    float yt = tbY[width*j0+i0];     // on suppose que le stockage tbX/tbY est ligne par ligne 
   
    int i1 = ((int) (xt*a2+b2) ) % width ;    // conversion en coordonn�es bitmap
    int j1 = ((int) (yt*a2+c2) ) % height;

    int ip1 = (i1+1) % width;
    int jp1 = (j1+1) % height;
    
    float z1 =   tb1[width*j1+i1];
    float z2 =   tb1[width*jp1+i1];
    float z3 =   tb1[width*jp1+ip1];
    float z4 =   tb1[width*j1+ip1];

    float dx =  xt-floorf(xt);
    float dy =  yt-floorf(yt); 

    float zp = z1+ dy*(z2-z1);
    float zq = z4+ dy*(z3-z4);
    float ZR = zp+ dx*(zq-zp);
            
    tb2[width*j+i] = ZR;    
   
}


